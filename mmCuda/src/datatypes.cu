#include "hip/hip_runtime.h"
#include "../include/datatypes.cuh"

__device__ unsigned long long counter5 = 0;

__global__ void print_counter5() { printf("Counter5: %ld \n", counter5); }

__host__ float h_Phi(const float &x, const float &y) {
  return atan2(y, x);
}

__host__ float h_R(const float &x, const float &y) {
  return sqrtf(x * x + y * y);
}

__host__ float h_Eta(const float &x, const float &y, const float &z) {
  float r = h_R(x, y);
  float r3 = sqrtf(r * r + z * z);
  float theta = acos(z / r3);

  return -log(tan(theta * 0.5));
}

__global__ void DeviceCalculateHitPairs(unsigned n_mod_pairs,
                                        ModuleDoublet *d_mod_doublets,
                                        unsigned *d_hit_offsets,
                                        unsigned *d_n_hit_pairs,
                                        unsigned *d_n_hits_a) {
  for (int i = blockIdx.x; i < n_mod_pairs; i += gridDim.x) {
    unsigned mod1 = d_mod_doublets[i].module_a;
    unsigned mod2 = d_mod_doublets[i].module_b;

    unsigned n_hits_mod1 = d_hit_offsets[mod1 + 1] - d_hit_offsets[mod1];
    unsigned n_hits_mod2 = d_hit_offsets[mod2 + 1] - d_hit_offsets[mod2];

    d_n_hits_a[i] = n_hits_mod1 + 1;
    d_n_hit_pairs[i] = n_hits_mod1 * n_hits_mod2;
  }
}

__global__ void DeviceCalculateHitTriplets(unsigned n_mod_triplets,
                                           ModuleTriplet *d_mod_triplets,
                                           unsigned *d_hit_offsets,
                                           unsigned long long *d_hit_triplets_offsets) {
  for (unsigned i = blockIdx.x; i < n_mod_triplets; i += gridDim.x) {
    unsigned mod1 = d_mod_triplets[i].module_a;
    unsigned mod2 = d_mod_triplets[i].module_b;
    unsigned mod3 = d_mod_triplets[i].module_c;

    unsigned n_hits_mod1 = d_hit_offsets[mod1 + 1] - d_hit_offsets[mod1];
    unsigned n_hits_mod2 = d_hit_offsets[mod2 + 1] - d_hit_offsets[mod2];
    unsigned n_hits_mod3 = d_hit_offsets[mod3 + 1] - d_hit_offsets[mod3];

    d_hit_triplets_offsets[i] = n_hits_mod1 * n_hits_mod2 * n_hits_mod3;
    atomicAdd(&counter5, n_hits_mod1 * n_hits_mod2 * n_hits_mod3);
  }
}

__global__ void DeviceInitHitPairs(unsigned n_mod_pairs,
                                   ModuleDoublet *d_mod_doublets,
                                   unsigned *d_hit_offsets,
                                   unsigned *hits_pairs_acc_offsets,
                                   unsigned *d_hits_a, unsigned *d_hits_b) {
  for (unsigned mod_idx = blockIdx.x; mod_idx < n_mod_pairs;
       mod_idx += gridDim.x) {
    unsigned mod1 = d_mod_doublets[mod_idx].module_a;
    unsigned mod2 = d_mod_doublets[mod_idx].module_b;

    unsigned n_hits_mod1 = d_hit_offsets[mod1 + 1] - d_hit_offsets[mod1];
    unsigned n_hits_mod2 = d_hit_offsets[mod2 + 1] - d_hit_offsets[mod2];

    unsigned hits_pairs_acc_offset = hits_pairs_acc_offsets[mod_idx];
    unsigned *hits_a = &d_hits_a[hits_pairs_acc_offset];
    unsigned *hits_b = &d_hits_b[hits_pairs_acc_offset];

    for (unsigned i = threadIdx.y; i < n_hits_mod1; i += blockDim.y) {
      unsigned *hits_a_row = &hits_a[i * n_hits_mod2];
      unsigned *hits_b_row = &hits_b[i * n_hits_mod2];
      for (unsigned j = threadIdx.x; j < n_hits_mod2; j += blockDim.x) {
        hits_a_row[j] = i;
        hits_b_row[j] = j;
      }
    }
  }
}

ModuleMap::ModuleMap(std::string mm_path_arg, std::string mm_pairs_path) {

  h_mod_doublets.reserve(ModuleMap::num_doublets());
  h_mod_triplets.reserve(ModuleMap::num_triplets());

  mm_path = mm_path_arg;

  std::string mm_line;
  std::string delim = " ";

  std::vector<unsigned> m_ids;
  m_ids.resize(2);
  std::vector<float> m_cuts;
  m_cuts.resize(10);
  std::vector<unsigned> m_pairs_sum;
  std::vector<unsigned> m_idx_b;
  m_pairs_sum.resize(num_modules());
  m_idx_b.reserve(num_doublets());
  /*********************************
  / Module doublets
  *********************************/
  std::ifstream mm_pairs_file(mm_pairs_path);
  if (mm_pairs_file.is_open()) {
    while (getline(mm_pairs_file, mm_line)) {

      auto start = 0U;
      auto end = mm_line.find(delim);

      for (int i = 0; i < 10; i++) {
        if (i < 2) {
          int m_id = std::stoi(mm_line.substr(start, end - start));
          m_ids[i] = m_id;
        } else if (i < 10) {
          float m_cut = std::stof(mm_line.substr(start, end - start));
          m_cuts[i] = m_cut;
        }
        start = end + delim.length();
        end = mm_line.find(delim, start);
      }

      ModuleDoublet m_doublet{m_ids[0],  m_ids[1],  m_cuts[3], m_cuts[2],
                              m_cuts[5], m_cuts[4], m_cuts[7], m_cuts[6],
                              m_cuts[9], m_cuts[8]};
      h_mod_doublets.push_back(m_doublet);
      m_pairs_sum[m_ids[0]] += 1;
      m_idx_b.push_back(m_ids[1]);
    }
    mm_pairs_file.close();
  } else {
    std::cout << "Unable to open file" << mm_pairs_path << std::endl;
  }
  std::cout << "Formed module doublets" << std::endl;

  // Prefix sum m_pairs_sum
  for (int i = 1; i < num_modules(); i++) {
    m_pairs_sum[i] += m_pairs_sum[i - 1];
  }
  // Add zero to the beginning
  m_pairs_sum.insert(m_pairs_sum.begin(), 0);

  m_ids.resize(5);
  m_cuts.resize(26);
  /*********************************
  / Module triplets
  *********************************/
  std::ifstream mm_file(mm_path);
  if (mm_file.is_open()) {
    while (getline(mm_file, mm_line)) {

      auto start = 0U;
      auto end = mm_line.find(delim);

      for (int i = 0; i < 26; i++) {
        if (i < 3) {
          int m_id = std::stoi(mm_line.substr(start, end - start));
          m_ids[i] = m_id;
        } else if (i == 24) {
          int m_id = std::stoi(mm_line.substr(
              start, end - start)); // Horrible hacky to get doublet pairs
          m_ids[3] = m_id;
        } else if (i == 25) {
          int m_id = std::stoi(mm_line.substr(start, end - start));
          m_ids[4] = m_id;
        } else {
          float m_cut = std::stof(mm_line.substr(start, end - start));
          m_cuts[i] = m_cut;
        }
        start = end + delim.length();
        end = mm_line.find(delim, start);
      }
      unsigned pair_i = findPairIndex(m_ids[0], m_ids[1], m_pairs_sum, m_idx_b);
      unsigned pair_j = findPairIndex(m_ids[1], m_ids[2], m_pairs_sum, m_idx_b);

      //std::cout << "i: " << m_ids[0] << " j: " << m_ids[1] << " k: " << m_ids[2] << " pair_i: " << pair_i << " pair_j: " << pair_j << std::endl;

      ModuleTriplet trip{
          m_ids[0],   m_ids[1],   m_ids[2],   pair_i,   pair_j,
          m_cuts[4],  m_cuts[5],  m_cuts[6],  m_cuts[7],  m_cuts[8],
          m_cuts[9],  m_cuts[10], m_cuts[11], m_cuts[12], m_cuts[13],
          m_cuts[14], m_cuts[15], m_cuts[16], m_cuts[17], m_cuts[18],
          m_cuts[19], m_cuts[20], m_cuts[21], m_cuts[22], m_cuts[23]};
      h_mod_triplets.push_back(trip);
    }
    mm_file.close();
  } else {
    std::cout << "Unable to open file" << mm_path << std::endl;
  }
  std::cout << "Formed module triplets" << std::endl;

  allocate_device_memory();

  CUDA_WARN(hipMemcpy(d_ptrs.doublets, h_mod_doublets.data(),
                       n_mod_pairs * sizeof(ModuleDoublet),
                       hipMemcpyHostToDevice));
  CUDA_WARN(hipMemcpy(d_ptrs.triplets, h_mod_triplets.data(),
                       n_mod_triplets * sizeof(ModuleTriplet),
                       hipMemcpyHostToDevice));
}

void ModuleMap::print_summary() {
  std::cout << std::string(50, '-') << std::endl;
  std::cout << "\tModule map summary" << std::endl;
  std::cout << std::string(50, '-') << std::endl;

  std::cout << "Module map taken from -" << std::endl;
  std::cout << mm_path << std::endl << std::endl;

  std::cout.imbue(std::locale(""));
  std::cout << std::fixed << std::showpoint << std::setprecision(3);

  std::cout << "# triplets: " << num_triplets() << std::endl;
  std::cout << "# doublets: " << num_doublets() << std::endl;

  std::cout << std::string(50, '-') << std::endl;
}

void ModuleMap::allocate_device_memory() {
  MemoryScheduler::allocate(&d_ptrs.doublets, n_mod_pairs);
  MemoryScheduler::allocate(&d_ptrs.triplets, n_mod_triplets);

  std::cout << "Allocated device memory for module map" << std::endl;
  unsigned long long int total_mem = 0;
  total_mem += n_mod_pairs * sizeof(ModuleDoublet);
  total_mem += n_mod_triplets * sizeof(ModuleTriplet);
  std::cout << "Total memory allocated: " << total_mem / 1e9 << " GB"
            << std::endl;
}

EventData::EventData(std::string event_path_arg, ModuleMap &mm,
                     unsigned int n_modules) {
  n_hits = 0;
  event_path = event_path_arg;

  // read the file and make the flatten triplets
  std::ifstream event_file(event_path);

  std::string hit_line;
  std::string delim = ",";

  if (event_file.is_open()) {

    // header (we ignore it for now)
    getline(event_file, hit_line);

    std::vector<std::string> col_entries;
    col_entries.reserve(27);
    h_hit_offsets.resize(
        ModuleMap::num_modules()); // Resize with number of modules
    float x;
    float y;
    while (getline(event_file, hit_line)) {

      // boost::split(col_entries, hit_line, boost::is_any_of(", "),
      // boost::token_compress_on);

      auto start = 0U;
      auto end = hit_line.find(delim);

      while (end != std::string::npos) {
        col_entries.push_back(hit_line.substr(start, end - start));

        start = end + delim.length();
        end = hit_line.find(delim, start);
      }
      col_entries.push_back(hit_line.substr(start, end - start));
      Hit hit;
      // Need to optimize this into one memory copy
      hit.x = std::stof(col_entries.at(2));
      hit.y = std::stof(col_entries.at(3));
      hit.z = std::stof(col_entries.at(4));
      hit.eta = h_Eta(hit.x, hit.y, hit.z);
      hit.phi = h_Phi(hit.x, hit.y);
      hit.r = h_R(hit.x, hit.y);
      h_hits.push_back(hit);
      unsigned mod = std::stoi(col_entries.back());
      if (mod >= ModuleMap::num_modules())
        std::cout << "Module ID is greater than number of modules" << std::endl;
      h_hit_inds.push_back(mod);
      h_hit_offsets[mod] += 1;
      col_entries.clear();
      n_hits += 1;
    } // end of while loop
  }
  event_file.close();

  allocate_device_memory_hits(n_hits);

  /*
      Sort Hits by module
  */
  unsigned *d_hit_inds;
  MemoryScheduler::allocate(&d_hit_inds, n_hits);
  CUDA_WARN(hipMemcpy(d_hit_inds, static_cast<unsigned *>(h_hit_inds.data()),
                       n_hits * sizeof(unsigned), hipMemcpyHostToDevice));

  Hit *d_hits;
  MemoryScheduler::allocate(&d_hits, n_hits);
  CUDA_WARN(hipMemcpy(d_hits, static_cast<Hit *>(h_hits.data()),
                       n_hits * sizeof(Hit), hipMemcpyHostToDevice));

  unsigned *d_hit_ind_out;
  MemoryScheduler::allocate(&d_hit_ind_out, n_hits);

  Sort(&d_hit_inds, &d_hits, &d_hit_ind_out, &d_ptrs.hits, n_hits);

  DEBUG(test_hit_sort(&d_hit_ind_out, n_hits));

  MemoryScheduler::free(&d_hit_inds);
  MemoryScheduler::free(&d_hits);
  MemoryScheduler::free(&d_hit_ind_out);

  /*
      Calculate Hit offsets
  */
  unsigned *d_hit_offsets;
  MemoryScheduler::allocate(&d_hit_offsets, ModuleMap::num_modules());
  CUDA_WARN(hipMemcpy(
      d_hit_offsets, static_cast<unsigned *>(h_hit_offsets.data()),
      ModuleMap::num_modules() * sizeof(unsigned), hipMemcpyHostToDevice));

  MemoryScheduler::memset(&d_ptrs.hit_offsets, ModuleMap::num_modules() + 1,
                          (unsigned)0);

  PrefixSum(&d_hit_offsets, &d_ptrs.hit_offsets, ModuleMap::num_modules());

  DEBUG(test_hit_offsets(h_hit_offsets, &d_ptrs.hit_offsets, ModuleMap::num_modules()));

  MemoryScheduler::free(&d_hit_offsets);

  /*
      Calculate Hit Pairs
  */
  unsigned *d_n_hit_pairs; // Number of hit pairs per doublet
  MemoryScheduler::allocate(&d_n_hit_pairs, ModuleMap::num_doublets());
  unsigned *d_n_hit_a; // Number of hits in first module of doublet
  MemoryScheduler::allocate(&d_n_hit_a, ModuleMap::num_doublets());

  CUDA_CHECK();

  // Strangely this is quicker with 65535 blocks rather than num_doublets
  // By like a factor of 5
  DeviceCalculateHitPairs<<<65535, 1>>>(ModuleMap::num_doublets(),
                                        *mm.d_doublets(), d_ptrs.hit_offsets,
                                        d_n_hit_pairs, d_n_hit_a);

  CUDA_CHECK();

  hipDeviceSynchronize();

  /*
    Calculate Hit Pairs Offsets Prefix Sum
  */
  MemoryScheduler::allocate(&d_ptrs.hit_pairs_offsets,
                            ModuleMap::num_doublets() + 1);

  MemoryScheduler::memset(&d_ptrs.hit_pairs_offsets,
                          ModuleMap::num_doublets() + 1, (unsigned)0);

  PrefixSum(&d_n_hit_pairs, &d_ptrs.hit_pairs_offsets,
            ModuleMap::num_doublets());

  DEBUG(test_hit_pairs_offsets(&d_n_hit_pairs, &d_ptrs.hit_pairs_offsets, ModuleMap::num_doublets()));

  MemoryScheduler::free(&d_n_hit_pairs);

  CUDA_CHECK();
  hipDeviceSynchronize();

  /*
      Calculate Hit Sum Offsets Prefix Sum
  */
  MemoryScheduler::allocate(&d_ptrs.hit_sum_offsets,
                            ModuleMap::num_doublets() + 1);

  MemoryScheduler::memset(&d_ptrs.hit_sum_offsets,
                          ModuleMap::num_doublets() + 1, (unsigned)0);

  PrefixSum(&d_n_hit_a, &d_ptrs.hit_sum_offsets,
            ModuleMap::num_doublets());

  DEBUG(test_hit_sum_offsets(&d_n_hit_a, &d_ptrs.hit_sum_offsets, ModuleMap::num_doublets()));

  MemoryScheduler::free(&d_n_hit_a);

  CUDA_CHECK();
  hipDeviceSynchronize();

  n_hit_pairs = getMax(&d_ptrs.hit_pairs_offsets, ModuleMap::num_doublets());

  hipDeviceSynchronize();


  MemoryScheduler::allocate(&d_ptrs.hit_pairs, n_hit_pairs);
  MemoryScheduler::allocate(&d_ptrs.hits_a, n_hit_pairs);
  MemoryScheduler::allocate(&d_ptrs.hits_b, n_hit_pairs);
  // TODO: When selection is added erduce the memory required for this.
  MemoryScheduler::allocate(&d_ptrs.hits_a_reduced, n_hit_pairs);
  MemoryScheduler::allocate(&d_ptrs.hits_b_reduced, n_hit_pairs);

  unsigned total = getMax(&d_ptrs.hit_sum_offsets, ModuleMap::num_doublets());

  hipDeviceSynchronize();

  MemoryScheduler::allocate(&d_ptrs.hit_sum, total);

  // MemoryScheduler::free(&d_total);

  dim3 grid_dim(65535);
  dim3 block_dim(32, 32);
  DeviceInitHitPairs<<<grid_dim, block_dim>>>(
      ModuleMap::num_doublets(), *mm.d_doublets(), d_ptrs.hit_offsets,
      d_ptrs.hit_pairs_offsets, d_ptrs.hits_a, d_ptrs.hits_b);

  MemoryScheduler::allocate(&d_ptrs.hit_module_sum, ModuleMap::num_doublets());

  unsigned long long *d_hit_triplets_offsets;
  MemoryScheduler::allocate(&d_hit_triplets_offsets, ModuleMap::num_triplets());
  MemoryScheduler::memset(&d_hit_triplets_offsets, ModuleMap::num_triplets(),
                          (unsigned long long)0);

  DeviceCalculateHitTriplets<<<65535, 1>>>(ModuleMap::num_triplets(),
                                           *mm.d_triplets(), d_ptrs.hit_offsets,
                                           d_hit_triplets_offsets);

  hipDeviceSynchronize();
  unsigned long long *d_hit_triplets_sum;
  MemoryScheduler::allocate(&d_hit_triplets_sum, ModuleMap::num_triplets() + 1);
  MemoryScheduler::memset(&d_hit_triplets_sum, ModuleMap::num_triplets() + 1,
                          (unsigned long long)0);

  PrefixSum(&d_hit_triplets_offsets, &d_hit_triplets_sum,
            ModuleMap::num_triplets());

  hipDeviceSynchronize();

  print_counter5<<<1,1>>>();

  // Get the total number of hit triplets
  n_hit_triplets = getMax(&d_hit_triplets_sum, ModuleMap::num_triplets());
  
  print_counter5<<<1,1>>>();
  hipDeviceSynchronize();
}

void EventData::print_summary() {
  std::cout << std::string(50, '-') << std::endl;
  std::cout << "\tEvent summary" << std::endl;
  std::cout << std::string(50, '-') << std::endl;

  std::cout << "Event taken from -" << std::endl;
  std::cout << event_path << std::endl << std::endl;

  std::cout.imbue(std::locale(""));
  std::cout << std::fixed << std::showpoint << std::setprecision(3);

  std::cout << "# hits: " << n_hits << std::endl;
  std::cout << "# hit pairs: " << n_hit_pairs << std::endl;
  std::cout << "# hit triplets: " << n_hit_triplets << std::endl;

  std::cout << std::string(50, '-') << std::endl;
}

void EventData::allocate_device_memory_hits(unsigned n_hits) {
  MemoryScheduler::allocate(&d_ptrs.hits, n_hits);
  MemoryScheduler::allocate(&d_ptrs.hit_offsets, ModuleMap::num_modules() + 1);
  MemoryScheduler::allocate(&d_ptrs.hit_module_sum, ModuleMap::num_doublets());
  MemoryScheduler::allocate(&d_ptrs.hit_module_offsets,
                            ModuleMap::num_doublets() + 1);
}

bool EventData::test_hit_sort(unsigned** d_hit_ind, unsigned n_hits) {
  unsigned *h_hit_ind = new unsigned[n_hits];

  CUDA_WARN(hipMemcpy(h_hit_ind, *d_hit_ind, n_hits * sizeof(unsigned),
                       hipMemcpyDeviceToHost));

  for (unsigned i = 0; i < n_hits - 1; i++) {
    if (h_hit_ind[i] > h_hit_ind[i + 1]) {
      std::cout << "\033[31m" << "Hit sort failed at " << i << "\033[0m" << std::endl;
      return false;
    }
  }

  delete[] h_hit_ind;
  std::cout << "\033[32m" << "Hit sort passed" << "\033[0m" << std::endl;
  return true;
}

bool EventData::test_hit_offsets(std::vector<unsigned>& h_hit_offsets, unsigned** d_hit_offsets, unsigned n_modules) {
  unsigned *h_hit_offsets_host = new unsigned[n_modules + 1];

  CUDA_WARN(hipMemcpy(h_hit_offsets_host, *d_hit_offsets, (n_modules + 1) * sizeof(unsigned),
                       hipMemcpyDeviceToHost));

  for (unsigned i = 0; i < n_modules; i++) {
    if (h_hit_offsets_host[i + 1] - h_hit_offsets_host[i] != h_hit_offsets[i]) {
      std::cout << "\033[31m" << "Hit offsets failed at " << i << "\033[0m" << std::endl;
      return false;
    }
  }

  delete[] h_hit_offsets_host;
  std::cout << "\033[32m" << "Hit offsets passed" << "\033[0m" << std::endl;
  return true;
}

bool EventData::test_hit_pairs_offsets(unsigned** d_n_hit_pairs, unsigned** d_hit_pairs_offsets, unsigned num_doublets) {
  unsigned *h_n_hit_pairs = new unsigned[num_doublets];
  unsigned *h_hit_pairs_offsets = new unsigned[num_doublets + 1];

  CUDA_WARN(hipMemcpy(h_n_hit_pairs, *d_n_hit_pairs, num_doublets * sizeof(unsigned),
                       hipMemcpyDeviceToHost));
  CUDA_WARN(hipMemcpy(h_hit_pairs_offsets, *d_hit_pairs_offsets, (num_doublets + 1) * sizeof(unsigned),
                       hipMemcpyDeviceToHost));

  for (unsigned i = 0; i < num_doublets; i++) {
    if (h_hit_pairs_offsets[i + 1] - h_hit_pairs_offsets[i] != h_n_hit_pairs[i]) {
      std::cout << "\033[31m" << "Hit pairs offsets failed at " << i << "\033[0m" << std::endl;
      return false;
    }
  }

  delete[] h_n_hit_pairs;
  delete[] h_hit_pairs_offsets;
  std::cout << "\033[32m" << "Hit pairs offsets passed" << "\033[0m" << std::endl;
  return true;
}

bool EventData::test_hit_sum_offsets(unsigned** d_n_hit_a, unsigned** d_hit_sum_offsets, unsigned num_doublets) {
  unsigned *h_n_hit_a = new unsigned[num_doublets];
  unsigned *h_hit_sum_offsets = new unsigned[num_doublets + 1];

  CUDA_WARN(hipMemcpy(h_n_hit_a, *d_n_hit_a, num_doublets * sizeof(unsigned),
                       hipMemcpyDeviceToHost));
  CUDA_WARN(hipMemcpy(h_hit_sum_offsets, *d_hit_sum_offsets, (num_doublets + 1) * sizeof(unsigned),
                       hipMemcpyDeviceToHost));

  for (unsigned i = 0; i < num_doublets; i++) {
    if (h_hit_sum_offsets[i + 1] - h_hit_sum_offsets[i] != h_n_hit_a[i]) {
      std::cout << "\033[31m" << "Hit sum offsets failed at " << i << "\033[0m" << std::endl;
      return false;
    }
  }

  delete[] h_n_hit_a;
  delete[] h_hit_sum_offsets;
  std::cout << "\033[32m" << "Hit sum offsets passed" << "\033[0m" << std::endl;
  return true;
}