#include "hip/hip_runtime.h"
#include "../include/triplet_finder.cuh"
#include "../include/datatypes.cuh"

#include <hip/hip_runtime_api.h>

#define N_MOD_PAIRS 509461

int main(int argc, char *argv[]) {

    std::string mm_path{"/srv01/agrp/shieldse/storage/ML/trackingData/transformed_data/module_map/df_MMTriplet_3hits_ptCut1GeV_woutSec_woutOC_90kevents_woutElectron.csv"};
    std::string mm_pairs_path{"/srv01/agrp/shieldse/storage/ML/trackingData/transformed_data/module_map/df_MMTriplet_3hits_ptCut1GeV_woutSec_woutOC_90kevents_woutElectron_pairs.csv"};
    ModuleMap mm = ModuleMap(mm_path, mm_pairs_path);
    mm.print_summary();
    // mm.cuda();

    unsigned int n_doublets = mm.len_doublets();

    
    std::string event_path{"/storage/agrp/nilotpal/tracking/transformed_data/events/event000000001-truth.csv"};
    EventData event = EventData(event_path, 18960);
    event.print_summary();
    
    const unsigned n_max_doublets = 1000;
    
    // Transfer module pairs onto the device
    // Only needs to be done once
    unsigned* h_mod_pairs = mm.doublets(); // Get from module map
    const unsigned n_mod_pairs = mm.len_doublets(); // Fix this to the correct number from module map
    unsigned* d_mod_pairs;
    const unsigned d_size = n_mod_pairs*sizeof(unsigned);
    hipMalloc((unsigned**)&d_mod_pairs, d_size); // Assign the memory on the device
    hipMemcpy(d_mod_pairs, h_mod_pairs, d_size, hipMemcpyHostToDevice); // Copy to device
    
    // Transfer module pairs offsets onto device
    // Only needs to be done once
    unsigned* h_mod_pairs_offsets = mm.offsets(); // Get from module map
    const unsigned n_mod = 18359; // Counted this from file
    unsigned* d_mod_pairs_offsets;
    const unsigned d_size_offsets = n_mod*sizeof(unsigned);
    hipMalloc((unsigned**)&d_mod_pairs_offsets, d_size_offsets); // Assign the memory on the device
    hipMemcpy(d_mod_pairs_offsets, h_mod_pairs_offsets, d_size_offsets, hipMemcpyHostToDevice);
    
    // If needed can perform a loop over events with streams
    // Need to figure our most efficient way to do asynchronous memory transfers
    // const unsigned n_streams = 1; // Maximum number that can run concurrently
    // Create the streams
    // hipStream_t streams[n_stream];
    // for (auto i : n_streams) {
    //     hipStreamCreate(&streams[i]);
    // }

    // Transfer atomics onto device
    // Only needs to be done once
    unsigned h_atomics_arr[N_MOD_PAIRS] = {}; // Initialise to zero
    unsigned* h_atomics = h_atomics_arr;
    unsigned* d_atomics;
    hipMalloc((unsigned**)&d_atomics, d_size);
    hipMemcpy(d_atomics, h_atomics, d_size, hipMemcpyHostToDevice);

    // Transfer triplets onto device
    unsigned* h_mod_triplets = mm.triplets();
    unsigned* d_mod_triplets;
    unsigned len_mod_triplets = mm.len_triplets();
    unsigned d_triplets_size = len_mod_triplets*sizeof(unsigned);
    hipMalloc((unsigned**)&d_mod_triplets, d_triplets_size);
    hipMemcpy(d_mod_triplets, h_mod_triplets, d_triplets_size, hipMemcpyHostToDevice);
    
    // Assign device memory for containers of hit indices, this stays on the device
    // and no memory transfers are needed.
    unsigned* d_hits_a_idx;
    unsigned* d_hits_b_idx;
    hipMalloc((unsigned**)&d_hits_a_idx, d_size * n_max_doublets);
    hipMalloc((unsigned**)&d_hits_b_idx, d_size * n_max_doublets);
    
    // Transfer hits onto device
    // Needs to be done for every event
    // If loop over events can be clever with assigning max memory needed
    Hit* h_hits = event.hits();
    Hit* d_hits;
    const unsigned n_hits = event.len(); // Get correct number
    const unsigned d_size_hits = n_hits * sizeof(Hit); // Fix for correct hit type
    hipMalloc((Hit**)&d_hits, d_size_hits); // Fix for correct hit type
    hipMemcpy(d_hits, h_hits, d_size_hits, hipMemcpyHostToDevice);
    
    // Transfer hit offsets onto device
    unsigned* h_hits_offsets = event.offsets();
    unsigned* d_hits_offsets;
    const unsigned d_size_hits_offsets = n_mod * sizeof(unsigned);
    hipMalloc((unsigned**)&d_hits_offsets, d_size_hits_offsets);
    hipMemcpy(d_hits_offsets, h_hits_offsets, d_size_hits_offsets, hipMemcpyHostToDevice);
    
    dim3 grid_dim(512,512);
    dim3 block_dim(32,32);
    // Call doublet finding kernal
    doublet_finding<<<grid_dim, block_dim>>>(
        n_mod,
        d_mod_pairs,
        d_mod_pairs_offsets,
        d_hits,
        d_hits_offsets,
        d_hits_a_idx,
        d_hits_b_idx,
        d_atomics,
        n_max_doublets
    );
    hipDeviceSynchronize();
    
    
    // Transfer doublets atomics back onto host
    hipMemcpy(h_atomics, d_atomics, d_size, hipMemcpyDeviceToHost);
    
    // Running cumulative sum of number of pairs that pass
    unsigned h_atomics_cumsum_arr[n_mod_pairs];
    unsigned* h_atomics_cumsum = h_atomics_cumsum_arr;
    PrefixSum(h_atomics, n_mod_pairs, h_atomics_cumsum);
    
    // Transer summed atomics back onto device
    hipMemcpy(d_atomics, h_atomics_cumsum, d_size, hipMemcpyHostToDevice);

    bool* d_doublet_acceptance;
    bool* d_hit_acceptance;
    hipMalloc((bool**)&d_doublet_acceptance, n_mod_pairs * sizeof(bool) * n_max_doublets);
    hipMalloc((bool**)&d_hit_acceptance, n_hits * sizeof(bool));


    unsigned n_mod_triplets = mm.num_triplets();
    
    // Call triplet finding kernal
    triplet_finding<<<256, block_dim>>>(
        n_mod_triplets,
        d_mod_triplets,
        d_mod_pairs,
        d_mod_pairs_offsets,
        d_hits,
        d_hits_offsets,
        d_hits_a_idx,
        d_hits_b_idx,
        d_atomics,
        d_doublet_acceptance,
        d_hit_acceptance,
        n_max_doublets
    );
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();  // add
    if (err != hipSuccess) std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl; // add
    hipProfilerStop();
    return 0;
    
    // Free up device memory
    // hipFree(d_mod_pairs);
    // hipFree(d_mod_pairs_offsets);

}